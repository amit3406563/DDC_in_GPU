// defining grid and block dimensions

dim3 block(512);
dim3 grid(((NX/512)+1)*(BATCH));

// doing multiplication with sine and cosine wave ....

hipEvent_t start, stop; // defining the event variables
        
HANDLE_ERROR(hipEventCreate(&start)); // creating events
HANDLE_ERROR(hipEventCreate(&stop));

HANDLE_ERROR( hipEventRecord( start, 0 )); // starting event
        
multi_sine_cosine<<<grid,block>>>(dev_sine, dev_cosine, dev_op_sine, dev_op_cosine, dev_input_host);
        
HANDLE_ERROR( hipEventRecord( stop, 0 )); // stopping the event
HANDLE_ERROR( hipEventSynchronize( stop)); // synchronizing the timings
    	
float elapsedtime; // defining elapsed time
HANDLE_ERROR( hipEventElapsedTime( &elapsedtime, start, stop )); // elapsed time

x += elapsedtime;

HANDLE_ERROR( hipEventDestroy( start )); // destroying events start and stop
HANDLE_ERROR( hipEventDestroy( stop ));


// doing LPF by using FIR filter using hamming window ....
      
HANDLE_ERROR(hipMemcpy(dev_lpf_hamming, lpf_hamming, windowLength * sizeof(float), hipMemcpyHostToDevice));
        
hipEvent_t start_conv, stop_conv; // defining the event variables
        
HANDLE_ERROR(hipEventCreate(&start_conv)); // creating events
HANDLE_ERROR(hipEventCreate(&stop_conv));
        
HANDLE_ERROR( hipEventRecord( start_conv, 0 )); // starting event
        
conv<<<grid,block>>>(dev_op_sine, dev_op_cosine, dev_op_sine_conv, dev_op_cosine_conv, dev_lpf_hamming, b, windowLength);
        
HANDLE_ERROR( hipEventRecord( stop_conv, 0 )); // stopping the event
HANDLE_ERROR( hipEventSynchronize( stop_conv)); // synchronizing the timings
    	
float elapsedtime_conv; // defining elapsed time
HANDLE_ERROR( hipEventElapsedTime( &elapsedtime_conv, start_conv, stop_conv )); // elapsed time
    	
y += elapsedtime_conv;
        
HANDLE_ERROR( hipEventDestroy( start_conv )); // destroying events start and stop
HANDLE_ERROR( hipEventDestroy( stop_conv ));


// make complex values ....
hipEvent_t start_comp, stop_comp; // defining the event variables
        
HANDLE_ERROR(hipEventCreate(&start_comp)); // creating events
HANDLE_ERROR(hipEventCreate(&stop_comp));
        
HANDLE_ERROR( hipEventRecord( start_comp, 0 )); // starting event
        
comp<<<grid,block>>>(dev_comp, dev_dec_sine_conv, dev_dec_cosine_conv, (c/DF));
        
HANDLE_ERROR( hipEventRecord( stop_comp, 0 )); // stopping the event
HANDLE_ERROR( hipEventSynchronize( stop_comp)); // synchronizing the timings
    	
float elapsedtime_comp; // defining elapsed time
HANDLE_ERROR( hipEventElapsedTime( &elapsedtime_comp, start_comp, stop_comp )); // elapsed time
    	
z += elapsedtime_comp;
        
HANDLE_ERROR( hipEventDestroy( start_comp )); // destroying events start and stop
HANDLE_ERROR( hipEventDestroy( stop_comp ));
