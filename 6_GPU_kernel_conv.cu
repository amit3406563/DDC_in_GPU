
#include <hip/hip_runtime.h>
// GPU kernel for convoluting sine and cosine multiplication data with filter coefficients with hamming window .... 

__global__ void conv(float *dev_op_sine, float *dev_op_cosine, float *dev_op_sine_conv, float *dev_op_cosine_conv, float *dev_lpf_hamming, int b, int windowLength){
  int i,k,l;
	
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  
  float temp1, temp2;
	for(i = idx; i < b; i+=stride){
    temp1 = 0;
		temp2 = 0;
				
		for(k = 0; k < windowLength; k++){	
			l = windowLength - k;
			temp1 +=  dev_op_sine[i+l] * dev_lpf_hamming[k];
      temp2 +=  dev_op_cosine[i+l] * dev_lpf_hamming[k];
    }
        
    dev_op_sine_conv[i] = temp1;
    dev_op_cosine_conv[i] = temp2;
  }
}
