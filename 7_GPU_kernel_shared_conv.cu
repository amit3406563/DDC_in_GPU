
#include <hip/hip_runtime.h>
// GPU kernel for convoluting sine and cosine multiplication data with filter coefficients with hamming window .... 

#define BLOCK_SIZE 512
#define WINDOW 500

__global__ void conv(float *dev_op_sine, float *dev_op_cosine, float *dev_op_sine_conv, float *dev_op_cosine_conv, float *dev_lpf_hamming, int b, int windowLength){
	int i,l;
       
	__shared__ float dats[BLOCK_SIZE];
	__shared__ float datc[BLOCK_SIZE];
  __shared__ float coeff[WINDOW];

  int threadid = threadIdx.x;	
	int idx = threadIdx.x + blockIdx.x * 256;
        
  float temp1, temp2;
        
	dats[threadid] = dev_op_sine[idx]; 
	datc[threadid] = dev_op_cosine[idx];
       
  if(threadid < windowLength){
    coeff[threadid] = dev_lpf_hamming[threadid];
  }
  
  __syncthreads();
       
  if(threadid < 256){
    temp1 = 0;
    temp2 = 0;
    
    for( i = 0; i < windowLength ; ++i){
      l = windowLength - i;
      
      temp1 += dats[threadid+l] * coeff[i];
      temp2 += datc[threadid+l] * coeff[i];
    }
    
    dev_op_sine_conv[idx] = temp1;
    dev_op_cosine_conv[idx] = temp2;
  }
}
