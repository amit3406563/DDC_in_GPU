#include "hip/hip_runtime.h"
// taking command line inputs and assigning them ....
if(argc != 9){
  fprintf(stderr, "USAGE: <windowLength> <sampFreq> <DF in int> <FFT_Size> <int> <sin/cos Freq> <file_name> <loops>\n");
  exit(1);
}
    
int windowLength = atoi(argv[1]);
float sampFreq = atof(argv[2]);
int DF = atoi(argv[3]);
float transFreq = (sampFreq/(2 * DF));
int NX = atoi(argv[4]);
int BATCH = atoi(argv[5]);
float f  = atof(argv[6]);
float s = atof(argv[8]);
int b = ((NX * BATCH) + windowLength);
int c = (NX * BATCH);
    
FILE *file = fopen((argv[7]), "r");

if (file == NULL){
  fprintf(stderr, "Can't open input file ! \n");
  exit(1);
}
