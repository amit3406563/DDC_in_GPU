// plan and memory for input FFT ....
hipfftHandle planip;
hipfftComplex *ip_fft;
hipMalloc((void**)&ip_fft, sizeof(hipfftComplex)*(NX/2+1)*BATCH);
	
//Create a 1D FFT plan
hipfftPlan1d(&planip, NX, HIPFFT_R2C, BATCH);
	
//allocating memory on host to store the copy of fft values calculated in device
Complex *ip_fft_result = (Complex *)malloc((BATCH)*(NX/2+1)*sizeof(Complex));
    

// plan and memory for output FFT ....
hipfftHandle plan;
hipfftComplex *data;
hipMalloc((void**)&data, sizeof(hipfftComplex)*(NX+1)*(BATCH/DF));
    	
//Create a 1D FFT plan
hipfftPlan1d(&plan, NX, HIPFFT_C2C, (BATCH/DF));
    
//allocating memory on host to store the copy of fft values calculated in device
Complex *result = (Complex *)malloc((BATCH/DF)*(NX)*sizeof(Complex)); 


// calculating R2C FFT in GPU using CUDA-C for plotting input FFT
hipfftExecR2C(planip, (hipfftReal*)dev_input_host, ip_fft);

// calculating C2C FFT in GPU using CUDA-C for plotting output FFT
hipfftExecC2C(plan, (hipfftComplex*)dev_comp, data,HIPFFT_FORWARD);


//Destroy the CUFFT plan
hipfftDestroy(plan);
hipFree(data);
hipfftDestroy(planip);
hipFree(ip_fft);
