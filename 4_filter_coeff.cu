#include "hip/hip_runtime.h"
// Create sinc function for filter - Low and High pass filters

enum filterType {LOW_PASS, HIGH_PASS};

float *create1TransSinc(int windowLength, float transFreq, float sampFreq, enum filterType type){
  int n;
  
  // Allocate memory for the window
  float *window = (float *) malloc(windowLength * sizeof(float));
	
  if (window == NULL){
    fprintf(stderr, "create1TransSinc: Could not allocate memory for window\n");
		return NULL;
	}

	if (type != LOW_PASS && type != HIGH_PASS) {
		fprintf(stderr, "create1TransSinc: Bad filter type, should be either LOW_PASS of HIGH_PASS\n");
		return NULL;
	}

	// Calculate the normalised transistion frequency. As transFreq should be
	// less than or equal to sampFreq / 2, ft should be less than 0.5
	float ft = transFreq / sampFreq;

	float m_2 = 0.5 * (windowLength-1);
	int halfLength = windowLength / 2;

	// Set centre tap, if present
	// This avoids a divide by zero
	if (2*halfLength != windowLength){
    float val = 2.0 * ft;

    // If we want a high pass filter, subtract sinc function from a dirac pulse
		if (type == HIGH_PASS) val = 1.0 - val;
    
    window[halfLength] = val;
  }
	
  else if (type == HIGH_PASS){
		fprintf(stderr, "create1TransSinc: For high pass filter, window length must be odd\n");
		return NULL;
	}

	// This has the effect of inverting all weight values
	if (type == HIGH_PASS) ft = -ft;

	// Calculate taps
	// Due to symmetry, only need to calculate half the window
	for (n=0 ; n<halfLength ; n++){
		float val = sin(2.0 * M_PI * ft * (n-m_2)) / (M_PI * (n-m_2));

		window[n] = val;
		window[windowLength-n-1] = val;
	}

	return window;
}


// create window function ....

enum windowType {HAMMING};

float *createWindow(float *in, float *out, int windowLength, enum windowType type){
  // If output buffer has not been allocated, allocate memory now
  if (out == NULL){
    out = (float *) malloc(windowLength * sizeof(float));
    if (out == NULL){
      fprintf(stderr, "Could not allocate memory for window\n");
      return NULL;
    }
  }

  int n;
  int m = windowLength - 1;
  int halfLength = windowLength / 2;

  // Calculate taps
  // Due to symmetry, only need to calculate half the window
  switch (type){
    case HAMMING:
      for (n=0 ; n<=halfLength ; n++){
        float val = 0.54 - 0.46 * cos(2.0 * M_PI * n / m);
        out[n] = val;
        out[windowLength-n-1] = val;
      }
      break;
  }

  // If input has been given, multiply with out
  if (in != NULL){
    for (n = 0 ; n < windowLength ; n++){
      out[n] *= in[n];
    }
  }
  
  return out;
}


// getting filter coefficients for a low pass filter using a sinc function and windowing using hamming window ....
lpf = create1TransSinc(windowLength, transFreq, sampFreq, LOW_PASS);
lpf_hamming = createWindow(lpf, NULL, windowLength, HAMMING);


// generating sine and cosine wave ....
float a = 1, wl;
float w = 2 * M_PI * f;
    
for(int n = 0; n < b; n++){
  wl = w * n;
  cosine[n] = a * cosf(wl/sampFreq);
  sine[n] = a * sinf(wl/sampFreq);
}
