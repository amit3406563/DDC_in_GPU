#include "hip/hip_runtime.h"
// GPU kernel function for multiplication of input signal with cosine and sine function .... 

__global__ void multi_sine_cosine(float *dev_sine, float *dev_cosine,  float *dev_op_sine, float *dev_op_cosine, float *dev_input_host){
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  
  float temp1, temp2;
  temp1 =  dev_input_host[idx] * dev_cosine[idx];
	temp2 = dev_input_host[idx] * dev_sine[idx];
		
  dev_op_cosine[idx] = temp1;
  dev_op_sine[idx] = temp2;
}


// GPU kernel function for assigning LPF values as real(sine multiplication) and complex(cosine multiplication) .... 

__global__ void comp(hipfftComplex *dev_comp, float *dev_op_sine_conv, float *dev_op_cosine_conv, int c){
  int i;
  
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  
  for(i = idx; i < c; i+=stride){
    dev_comp[i].x = dev_op_cosine_conv[i];
    dev_comp[i].y = -1*dev_op_sine_conv[i];
  }
}
