// on GPU ....
float *dev_input_host;
HANDLE_ERROR( hipMalloc((float **)&dev_input_host, b * sizeof(float)));
    
float *dev_sine;
float *dev_cosine;
float *dev_op_cosine;
float *dev_op_sine;
        
HANDLE_ERROR( hipMalloc((float **)&dev_sine, b * sizeof(float)));
HANDLE_ERROR( hipMalloc((float **)&dev_cosine, b * sizeof(float)));
HANDLE_ERROR( hipMalloc((float **)&dev_op_sine, b * sizeof(float)));
HANDLE_ERROR( hipMalloc((float **)&dev_op_cosine, b * sizeof(float)));
    
float *dev_op_cosine_conv;
float *dev_op_sine_conv;
float *dev_lpf_hamming;
        
HANDLE_ERROR( hipMalloc((float **)&dev_op_sine_conv, b * sizeof(float)));
HANDLE_ERROR( hipMalloc((float **)&dev_op_cosine_conv, b * sizeof(float)));
HANDLE_ERROR( hipMalloc((float **)&dev_lpf_hamming, windowLength * sizeof(float)));	
    
float *dev_dec_sine_conv;
float *dev_dec_cosine_conv;
        
HANDLE_ERROR( hipMalloc((float **)&dev_dec_sine_conv, (c/DF) * sizeof(float)));
HANDLE_ERROR( hipMalloc((float **)&dev_dec_cosine_conv, (c/DF) * sizeof(float)));
    
Complex *dev_comp;
        
HANDLE_ERROR(hipMalloc ((Complex **)&dev_comp, (c/DF) * sizeof(Complex)));

// freeing all memory on device (GPU) ....
hipFree(dev_input_host);
hipFree(dev_cosine);
hipFree(dev_sine);
hipFree(dev_op_sine);
hipFree(dev_op_cosine);
hipFree(dev_op_sine_conv);
hipFree(dev_op_cosine_conv);
hipFree(dev_comp);
hipFree(dev_lpf_hamming);
hipFree(dev_dec_sine_conv);
hipFree(dev_dec_cosine_conv);
